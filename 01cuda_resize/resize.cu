#include "hip/hip_runtime.h"
#include <thrust/fill.h>
#include "resize.h"

// resize, crop, norm
// sample : Nearest
__global__ void preprocess_nearest_kernel(const uchar* __restrict__ src_dev, 
                                    float* __restrict__ dst_dev, int src_row_step, 
                                    int dst_row_step, int src_img_step, int dst_img_step,
                                    int src_h, int src_w, float radio_h, float radio_w, 
                                    float offset_h, float offset_w, triplet mean, triplet std){
	int i = blockIdx.x;
	int j = blockIdx.y;
    int k = threadIdx.x;

	int pX = (int) roundf((i / radio_h) + offset_h);
	int pY = (int) roundf((j / radio_w) + offset_w);
 
	if (pX < src_h && pX >= 0 && pY < src_w && pY >= 0){
        int s1 = k * src_img_step + 0 * src_img_step / 3 + pX * src_row_step + pY;
        int s2 = k * src_img_step + 1 * src_img_step / 3 + pX * src_row_step + pY;
        int s3 = k * src_img_step + 2 * src_img_step / 3 + pX * src_row_step + pY;

        int d1 = k * dst_img_step + 0 * dst_img_step / 3 + i * dst_row_step + j;
        int d2 = k * dst_img_step + 1 * dst_img_step / 3 + i * dst_row_step + j;
        int d3 = k * dst_img_step + 2 * dst_img_step / 3 + i * dst_row_step + j;

		*(dst_dev + d1) = ((float)*(src_dev + s1) - mean.x) / std.x;
		*(dst_dev + d2) = ((float)*(src_dev + s2) - mean.y) / std.y;
		*(dst_dev + d3) = ((float)*(src_dev + s3) - mean.z) / std.z;
	}
}


int preprocess(const uchar* src_imgs, float* dst_imgs, int n_img, int src_img_h,
                int src_img_w, int dst_img_h, int dst_img_w, float resize_radio_h, 
                float resize_radio_w, int crop_h, int crop_w, triplet mean, 
                triplet std, Sampler sample){
    /*
    src_imgs : 6 * 3 * src_img_h * src_img_w
    dst_imgs : 6 * 3 * dst_img_h * dst_img_w
    crop_h : resize后的图像，纵向自上裁剪范围
    crop_w : 为0
    */


    int src_row_step = src_img_w;
    int dst_row_step = dst_img_w;
    int src_img_step = src_img_w * src_img_h * 3;
    int dst_img_step = dst_img_w * dst_img_h * 3;

    float offset_h = crop_h / resize_radio_h;
    float offset_w = crop_w / resize_radio_w;

    dim3 grid(dst_img_h, dst_img_w);
    dim3 block;
    
    // printf("sampler : nearest\n");
    block = dim3(n_img);
    preprocess_nearest_kernel<<<grid, block>>>(src_imgs, dst_imgs, src_row_step, dst_row_step, 
                    src_img_step, dst_img_step, src_img_h, src_img_w, resize_radio_h,
                    resize_radio_w, offset_h, offset_w, mean, std);

    return EXIT_SUCCESS;
}
